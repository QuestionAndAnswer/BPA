#include "hip/hip_runtime.h"
/**
 * Author: rodrigo
 * 2015
 */
#include "CudaUtil.h"
#include <hip/hip_runtime.h>
#include <ostream>
#include <iostream>

#define N	20
#define cudaCheckErrors(msg) \
	do { \
		hipError_t __err = hipGetLastError(); \
		if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
			msg, hipGetErrorString(__err), \
			__FILE__, __LINE__); \
			fprintf(stderr, "*** FAILED - ABORTING\n"); \
			exit(1); \
		} \
	} while (0)

struct BallCenter
{
	float cx, cy, cz;
	float radius;
};

struct Point
{
	float x, y, z, w;
	float nx, ny, nz, nw;
	float c;
	float fill[3];
};

std::ostream &operator<<(std::ostream &_stream, const BallCenter &_center)
{
	_stream << "c=(" << _center.cx << ", " << _center.cy << ", " << _center.cz << ") / r=" << _center.radius;
	return _stream;
}

__global__ void calculateBalls(Point *_points, BallCenter *_balls)
{
	for (int i = 0; i < N; i++)
	{
		_balls[i].cx = i;
		_balls[i].cy = i;
		_balls[i].cz = i;
		_balls[i].radius = _points[i].x;
	}
}

void CudaUtil::calculateBallCenters(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud)
{
	size_t cloudSize = _cloud->size();

	// Alloc memory on the device and copy cloud data to it
	Point *devPoints;
	hipMalloc((void **) &devPoints, sizeof(pcl::PointNormal) * cloudSize);
	cudaCheckErrors("hipMalloc 1 failed");

	hipMemcpy(devPoints, &_cloud->points[0], sizeof(pcl::PointNormal) * cloudSize, hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy to dev failed");

	// Alloc memory for the results
	BallCenter *devBalls;
	hipMalloc((void **) &devBalls, sizeof(BallCenter) * cloudSize * cloudSize * cloudSize);
	cudaCheckErrors("hipMalloc 2 failed");

	calculateBalls<<<1, 1>>>(devPoints, devBalls);

	// Alloc memory on host
	BallCenter *balls = (BallCenter*) malloc(sizeof(BallCenter) * cloudSize * cloudSize * cloudSize);
	hipMemcpy(balls, devBalls, sizeof(BallCenter) * cloudSize * cloudSize * cloudSize, hipMemcpyDeviceToHost);
	cudaCheckErrors("hipMemcpy to host failed");

	for (int i = 0; i < N; i++)
		std::cout << balls[i] << std::endl;

	int x = 0;
}
