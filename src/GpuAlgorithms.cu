#include "hip/hip_runtime.h"
/**
 * Author: rodrigo
 * 2015
 */
#include "GpuAlgorithms.h"
#include <ostream>
#include <iostream>
#include "GpuUtils.h"

#define EPSILON	1E-10

// Pointers to memory in device
Point *devPoints = NULL;
bool *devNotUsed = NULL;
BallCenter *auxPtr = NULL;

// Global variable in device
__device__ int devFound;
__device__ BallCenter *devCenter;
__device__ Point *devPointDbg;

std::ostream &operator<<(std::ostream &_stream, const BallCenter &_center)
{
	_stream << "c=(" << _center.cx << ", " << _center.cy << ", " << _center.cz << ") / (" << _center.idx0 << ", " << _center.idx1 << ", " << _center.idx2 << ")";
	return _stream;
}

void GpuAlgorithms::allocPoints(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud)
{
	Point *hostPoints = (Point *) &_cloud->points[0];
	GpuUtils::createInDev<Point>(&devPoints, hostPoints, _cloud->size());
}

void GpuAlgorithms::allocUsed(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud, const bool* _notUsed)
{
	GpuUtils::createInDev<bool>(&devNotUsed, _notUsed, _cloud->size());
}

__global__ void searchCloserPoints(const int _target, const Point *_points, const int _pointNumber, const double _searchRadius, const int _pointsPerThread, bool *_selected)
{
	int startIdx = (blockIdx.x * blockDim.x + threadIdx.x) * _pointsPerThread;
	double sqrRadius = _searchRadius * _searchRadius;

	for (int i = startIdx; i < startIdx + _pointsPerThread && i < _pointNumber; i++)
	{
		_selected[i] = _points[_target].sqrDist(_points[i]) < sqrRadius;
	}
}

bool GpuAlgorithms::radiusSearch(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud, const int _target, double _radius, std::vector<int> &_idxs)
{
	int blocks = 10;
	int threads = 256;
	size_t cloudSize = _cloud->size();

	// Copy points to device
	if (devPoints == NULL)
		allocPoints(_cloud);

	// Array to store points within radius
	bool *devSelected;
	hipMalloc((void **) &devSelected, sizeof(bool) * cloudSize);
	checkErrors("hipMalloc selected failed");

	// Calculate adequate number of blocks and threads
	while (cloudSize / blocks < 2)
		blocks /= 2;
	int pointsPerBlock = ceil((double) cloudSize / blocks);

	while (pointsPerBlock / threads < 1)
		threads /= 2;
	int pointsPerThread = ceil((double) pointsPerBlock / threads);

	// Execute kernel
	searchCloserPoints<<<blocks, threads>>>(_target, devPoints, cloudSize, _radius, pointsPerThread, devSelected);

	// Copy data to host
	bool *selected = (bool *) calloc(cloudSize, sizeof(bool));
	hipMemcpy(selected, devSelected, sizeof(bool) * cloudSize, hipMemcpyDeviceToHost);
	checkErrors("hipMemcpy selected failed");
	//hipFree(devSelected);
	//checkErrors("hipFree selected failed");

	for (size_t i = 0; i < cloudSize; i++)
		if (selected[i])
			_idxs.push_back(i);

	free(selected);

	return true;
}

///////////////////////////////
__device__ bool isOriented(const Point *_normal, const Point *_p0, const Point *_p1, const Point *_p2)
{
	int count = 0;
	count = _p0->normalDot(*_normal) < 0 ? count + 1 : count;
	count = _p1->normalDot(*_normal) < 0 ? count + 1 : count;
	count = _p2->normalDot(*_normal) < 0 ? count + 1 : count;

	return count <= 1;
}

__device__ float getCircumscribedCircle(const Point &_p0, const Point &_p1, const Point &_p2, BallCenter &_center)
{
	Point d10 = _p1 - _p0;
	Point d20 = _p2 - _p0;
	Point d01 = _p0 - _p1;
	Point d12 = _p1 - _p2;
	Point d21 = _p2 - _p1;
	Point d02 = _p0 - _p2;

	float norm01 = d01.norm();
	float norm12 = d12.norm();
	float norm02 = d02.norm();

	float norm01C12 = d01.cross(d12).norm();

	float alpha = (norm12 * norm12 * d01.dot(d02)) / (2 * norm01C12 * norm01C12);
	float beta = (norm02 * norm02 * d10.dot(d12)) / (2 * norm01C12 * norm01C12);
	float gamma = (norm01 * norm01 * d20.dot(d21)) / (2 * norm01C12 * norm01C12);

	Point circumscribedCircleCenter = (_p0 * alpha) + (_p1 * beta) + (_p2 * gamma);
	float circumscribedCircleRadius = (norm01 * norm12 * norm02) / (2 * norm01C12);

	_center.cx = circumscribedCircleCenter.x;
	_center.cy = circumscribedCircleCenter.y;
	_center.cz = circumscribedCircleCenter.z;

	return circumscribedCircleRadius;
}

__device__ bool getBallCenter(const Point *_point0, const Point *_point1, const Point *_point2, const float _ballRadius, BallCenter *_center)
{
	bool status = false;
	_center->isValid = false;

	// Local pointers
	const Point *p0 = _point0;
	const Point *p1 = _point1;
	const Point *p2 = _point2;

	Point v10 = p1->operator -(*p0);
	Point v20 = p2->operator -(*p0);
	Point normal = v10.cross(v20);

	// Calculate ball center only if points are not collinear
	if (normal.norm() > EPSILON)
	{
		// Normalize to avoid precision errors while checking the orientation
		normal.normalize();
		if (!isOriented(&normal, p0, p1, p2))
		{
			// Wrong orientation, swap vertices to get a CCW oriented triangle so face's normal pointing upwards
			int aux = _center->idx0;
			_center->idx0 = _center->idx1;
			_center->idx1 = aux;

			p0 = _point1;
			p1 = _point0;

			v10 = p1->operator -(*p0);
			v20 = p2->operator -(*p0);
			normal = v10.cross(v20);
			normal.normalize();
		}

		float circleRadius = getCircumscribedCircle(*p0, *p1, *p2, *_center);
		float squaredDistance = _ballRadius * _ballRadius - circleRadius * circleRadius;

		if (squaredDistance > 0)
		{
			float distance = sqrt(fabs(squaredDistance));
			_center->add(normal * distance);
			_center->isValid = true;
			status = true;
		}
	}

	return status;
}

__device__ bool isEmpty(const BallCenter *_center, const Point *_points, const int _pointNumber, const float _ballRadius)
{
	for (int i = 0; i < _pointNumber; i++)
	{
		if ((i == _center->idx0) || (i == _center->idx1) || (i == _center->idx2))
			continue;

		if (_center->dist(_points[i]) >= _ballRadius)
			continue;

		return false;
	}

	return true;
}

__global__ void checkForSeeds(const Point *_points, const int _pointNumber, const int *_neighbors, const int _neighborsSize, const bool *_notUsed, const int _index0, const float _ballRadius)
{
	int startIdx = threadIdx.x;
	int endIdx = startIdx + 1;

	for (int j = startIdx; j < endIdx && j < _neighborsSize; j++)
	{
		if (devFound == 0)
		{
			int index1 = _neighbors[j];

			// Skip invalid combinations
			if (index1 == _index0 || !_notUsed[index1])
				continue;

			for (size_t k = 0; k < _neighborsSize && devFound == 0; k++)
			{
				int index2 = _neighbors[k];

				// Skip invalid combinations
				if (index1 == index2 || index2 == _index0 || !_notUsed[index2])
					continue;

				BallCenter center(_index0, index1, index2);
				if (getBallCenter(&_points[_index0], &_points[index1], &_points[index2], _ballRadius, &center))
				{
					//std::vector<int> neighborhood = getNeighbors(ballCenter, _ballRadius);
					if (isEmpty(&center, _points, _pointNumber, _ballRadius))
					{
						if (devFound == 0)
						{
							atomicExch(&devFound, 1);
							devCenter->cx = center.cx;
							devCenter->cy = center.cy;
							devCenter->cz = center.cz;
							devCenter->idx0 = center.idx0;
							devCenter->idx1 = center.idx1;
							devCenter->idx2 = center.idx2;
							devCenter->isValid = center.isValid;
							break;
						}
					}
				}
			}
		}
	}
}

BallCenter GpuAlgorithms::findSeed(const pcl::PointCloud<pcl::PointNormal>::Ptr &_cloud, const std::vector<int> &_neighbors, const bool *_notUsed, const int _index0, const float _ballRadius)
{
	int blocks = 1;
	int threads = _neighbors.size();
	size_t cloudSize = _cloud->size();

	// Prepare memory buffers
	if (devPoints == NULL)
		allocPoints(_cloud);
	if (devNotUsed == NULL)
		allocUsed(_cloud, _notUsed);

	// Copy not used data to dev
	size_t notUsedBytes = sizeof(bool) * _cloud->size();
	GpuUtils::setData<bool>(&devNotUsed, _notUsed, _cloud->size());

	// Create and prepare buffer with neighbors indices
	int *devNeighbors;
	GpuUtils::createInDev<int>(&devNeighbors, &_neighbors[0], _neighbors.size());

	// Prepare global variable 'devFound'
	int found = 0;
	GpuUtils::setSymbol<int>(devFound, &found);

	// Prepare global variable 'devFoundCenter'
	BallCenter center = BallCenter();
	if (auxPtr == NULL)
		GpuUtils::createInDev<BallCenter>(&auxPtr, &center, 1);
	GpuUtils::setData<BallCenter>(&auxPtr, &center, 1);
	GpuUtils::setSymbol<BallCenter *>(devCenter, &auxPtr);

	// Execute kernel
	checkForSeeds<<<blocks, threads>>>(devPoints, _cloud->size(), devNeighbors, _neighbors.size(), devNotUsed, _index0, _ballRadius);

	// Retrieve found status (just for debug)
	//hipMemcpyFromSymbol(&found, HIP_SYMBOL(devFound), sizeof(int));
	//checkErrors("hipMemcpyFromSymbol failed");

	// Retrieve results
	GpuUtils::getData<BallCenter>(&center, auxPtr, 1);

	// Free allocated memory
	hipFree(devNeighbors);
	checkErrors("hipFree devNeighbors failed");

	return center;
}
